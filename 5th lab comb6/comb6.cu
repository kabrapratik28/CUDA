#include<stdio.h>
#include<hip/hip_runtime.h>
//#include<conio.h>
#include<malloc.h>
double *a,*b,*c;
double *ga,*gb,*gc;
int numOfBlocks; 
int blocksize=256;
float et;
int sizerowa , sizeclma;

__device__ float comb6 (int n , int k)
{
  int i = 1 ; 
  float t = 1.0 ; 
  if (k<(n-k))
    {
      for(i=n;i>=(n-k+1);i--)
	{
	  t = t * i /(n-i+1); 
	}
    }

  else 
    {
      for (i=n; i>=(k+1); i--  )
	{
	  t = t * i / (n-i+1);
	}
    }
  return t ; 
}


__global__ void combcal (double *d_p,int kk, int blocksize)
{
int u,v;
u= ((blockIdx.x*blocksize)+threadIdx.x)/(kk);
v = (( blockIdx.x * blocksize )+threadIdx.x)%(kk); 

if(u>=v)
{
	d_p[threadIdx.x + (blocksize * blockIdx.x)] = comb6(u,v);
}

}


int main()
{

printf("Enter user matrix siz\n");
scanf("%d", &sizerowa);

sizeclma = sizerowa;

a=(double *)malloc((sizerowa* sizeclma)*sizeof(double));
//b=(int*)malloc((sizerowa * sizeclma)*sizeof(int));
//c=(int*)malloc((sizerowa * sizeclma)*sizeof(int));

//printf("%d %d \n",sizerowa, sizeclma);
int i=0; 

/*
 for(i=0;i<( sizerowa* sizeclma);i++)
{
*(a+i)=i;
}


 for(i=0;i<( sizerowa* sizeclma);i++)
 {
 *(b+i)=i;
 }
*/

numOfBlocks=( sizerowa* sizeclma) /blocksize;
if((sizerowa* sizeclma)%blocksize>0) numOfBlocks++;

hipMalloc((void**)&ga,( sizerowa* sizeclma) *sizeof(double));
//cudaMalloc((void**)&gb,( sizerowa* sizeclma)*sizeof(int));
//cudaMalloc((void**)&gc,( sizerowa* sizeclma)*sizeof(int));


//cudaMemcpy(ga,a,( sizerowa* sizeclma)*sizeof(int),cudaMemcpyHostToDevice);
//cudaMemcpy(gb,b, sizerowa* sizeclma *sizeof(int),cudaMemcpyHostToDevice);

//printf("%d %d \n",sizerowa, sizeclma);

		 hipEvent_t start,stop;
		 hipEventCreate(&start);
		 hipEventCreate(&stop);
		 hipEventRecord(start,0);
			combcal<<<numOfBlocks,blocksize>>>(ga,sizerowa,blocksize);

				hipEventRecord(stop,0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&et,start,stop);
				hipEventDestroy(start);
				hipEventDestroy(stop);
				hipDeviceSynchronize();

				hipMemcpy(a,ga,( sizerowa* sizeclma)*sizeof(double),hipMemcpyDeviceToHost);
				
				for(int jj=0; jj< sizerowa ; jj++)
				{
					for(int oo=0 ; oo < sizerowa ; oo++)
					{
						printf("%f  ",*(a+(sizerowa*jj)+oo));
					}
					printf("\n");
				}				

				printf(" parallel %f\n",et);


return 0 ; 
}



