#include<stdio.h>
#include<hip/hip_runtime.h>
//#include<conio.h>
#include<malloc.h>
int *a,*b,*c;
int *ga,*gb,*gc;
int numOfBlocks; 
int blocksize=256;
float et;
int sizerowa , sizeclma;

__global__ void MatrixMatrixadd (int* ga,int *gb,int* gc,int size)
{

int i=threadIdx.x + (blockIdx.x*blockDim.x);

  if(i<size)
{ 
  gc[i] =ga[i]+gb[i];
}

}


int main()
{

printf("Enter user matrix siz\n");
scanf("%d %d", &sizerowa ,  &sizeclma);

a=(int*)malloc((sizerowa* sizeclma)*sizeof(int));


int i=0; 



hipMalloc((void**)&ga,( sizerowa* sizeclma) *sizeof(int));
int *p ; 
hipHostAlloc(&p ,(sizerowa*sizeclma)*sizeof(int),hipHostMallocPortable );



		 hipEvent_t start,stop;
		 hipEventCreate(&start);
		 hipEventCreate(&stop);
		 hipEventRecord(start,0);

		 hipMemcpy(ga,a,( sizerowa* sizeclma)*sizeof(int),hipMemcpyHostToDevice);
				
				hipEventRecord(stop,0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&et,start,stop);
				hipEventDestroy(start);
				hipEventDestroy(stop);
				hipDeviceSynchronize();

				
				printf("without page locked parallel %f\n",et);




				//cudaEvent_t start,stop;
		 hipEventCreate(&start);
		 hipEventCreate(&stop);
		 hipEventRecord(start,0);

		 hipMemcpy(ga,p,( sizerowa* sizeclma)*sizeof(int),hipMemcpyHostToDevice);
				
				hipEventRecord(stop,0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&et,start,stop);
				hipEventDestroy(start);
				hipEventDestroy(stop);
				hipDeviceSynchronize();

				
				printf("pagelocked parallel %f\n",et);



return 0 ; 
}



