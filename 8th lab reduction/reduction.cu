//nvcc -arch=sm_20 reduction.cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<malloc.h>

int *a,*c;
int *ga,*gc;
int sizerowa;
int numOfBlocks; 
int blocksize=256;
float et;
__global__ void VectorMatrix(int* ga,int* gc,int sizerowa)
{
extern  __shared__ int s[];
int i=threadIdx.x;


//copy max in s[i]
 s[i]=999999; 
 __syncthreads();

//copy the values from d_var in s[i] 
 if (i<sizerowa)
    s[i]=ga[i];
 __syncthreads();

printf ("%d  ",s[i]);

//Do the reduction

 if (blockDim.x>=256)
 {
	if(i<128)
	{
		if(s[i]>s[i+128])
		{
			s[i]= s[i+128];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=128)
 {
	if(i<64)
	{
		if(s[i]>s[i+64])
		{
			s[i]= s[i+64];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=64)
 {
	if(i<32)
	{
		if(s[i]>s[i+32])
		{
			s[i]= s[i+32];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=32)
 {
	if(i<16)
	{
		if(s[i]>s[i+16])
		{
			s[i]= s[i+16];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=16)
 {
	if(i<8)
	{
		if(s[i]>s[i+8])
		{
			s[i]= s[i+8];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=8)
 {
	if(i<4)
	{
		if(s[i]>s[i+4])
		{
			s[i]= s[i+4];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=4)
 {
	if(i<2)
	{
		if(s[i]>s[i+2])
		{
			s[i]= s[i+2];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=2)
 {
	if(i<1)
	{
		if(s[i]>s[i+1])
		{
			s[i]= s[i+1];
		}
 	}

	__syncthreads( );
 }
 
//Thread zero will store minimum of this block in d_Min
if(i==0)
{
	*(gc+0)=s[0];
	printf("\nparallel %d\n",s[0]);
}


}


int main()
{

printf("Enter user matrix siz\n");
scanf("%d", &sizerowa );

a=(int*)malloc(sizerowa*sizeof(int));
c=(int*)malloc((1)*sizeof(int));


int i=0; 
 
 for(i=0;i<sizerowa;i++)
{
*(a+i)=i+2;
}


numOfBlocks=sizerowa/blocksize;
if(sizerowa%blocksize>0) numOfBlocks++;

hipMalloc((void**)&ga,sizerowa*sizeof(int));
hipMalloc((void**)&gc,1*sizeof(int));


hipMemcpy(ga,a,sizerowa*sizeof(int),hipMemcpyHostToDevice);


	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
			VectorMatrix<<<numOfBlocks,blocksize,sizerowa*sizeof(int)>>>(ga,gc,sizerowa);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&et,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
		hipDeviceSynchronize();

		hipMemcpy(c,gc,1*sizeof(int),hipMemcpyDeviceToHost);
			
	printf(" parallel %f\n",et);

printf("%d",*c);
return 0 ; 
}

