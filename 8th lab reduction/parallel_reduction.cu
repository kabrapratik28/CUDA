
//nvcc -arch=sm_20 reduction.cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<malloc.h>

int *a,*c;
int *ga,*gc;
int sizerowa;
int numOfBlocks; 
int blocksize=256;
float et;
__global__ void VectorMatrix(int* ga,int* gc,int sizerowa,int *secondtime,int count_take_first_second)
{
extern  __shared__ int s[];
int i=threadIdx.x;


//copy max in s[i]
 s[i]=99999; 
 __syncthreads();

//copy the values from d_var in s[i] 
int index = threadIdx.x+(blockDim.x*blockIdx.x);
 if (index<sizerowa)
    if (count_take_first_second==0)
    {
	s[i]=ga[index];
    }
    else 
    {
	s[i]=secondtime[index];
    }
 __syncthreads();

//printf ("%d  ",s[i]);

//Do the reduction

 if (blockDim.x>=256)
 {
	if(i<128)
	{
		if(s[i]>s[i+128])
		{
			s[i]= s[i+128];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=128)
 {
	if(i<64)
	{
		if(s[i]>s[i+64])
		{
			s[i]= s[i+64];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=64)
 {
	if(i<32)
	{
		if(s[i]>s[i+32])
		{
			s[i]= s[i+32];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=32)
 {
	if(i<16)
	{
		if(s[i]>s[i+16])
		{
			s[i]= s[i+16];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=16)
 {
	if(i<8)
	{
		if(s[i]>s[i+8])
		{
			s[i]= s[i+8];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=8)
 {
	if(i<4)
	{
		if(s[i]>s[i+4])
		{
			s[i]= s[i+4];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=4)
 {
	if(i<2)
	{
		if(s[i]>s[i+2])
		{
			s[i]= s[i+2];
		}
 	}

	__syncthreads( );
 }

 if (blockDim.x>=2)
 {
	if(i<1)
	{
		if(s[i]>s[i+1])
		{

			s[i]= s[i+1];
		}
 	}

	__syncthreads( );
 }
 
//Thread zero will store minimum of this block in d_Min
if(i==0)
{
	*(gc+blockIdx.x)=s[0];
//	printf("\nblock minimum value  %d\n",s[0]);
}


}


int main()
{

printf("Enter user matrix siz\n");
scanf("%d", &sizerowa );

  
a=(int*)malloc(sizerowa*sizeof(int));
c=(int*)malloc((sizerowa)*sizeof(int));


int i=0; 
int flag = 0 ;
int current_size = sizerowa; 
int count_first_second = 0 ; 
 for(i=0;i<sizerowa;i++)
{
	*(a+i)=i+2;
}
//test only for size greater than 1000
//*(a+290)= -5 ; 
//*(a+800)=-6;

numOfBlocks=sizerowa/blocksize;
if(sizerowa%blocksize>0) numOfBlocks++;

hipMalloc((void**)&ga,sizerowa*sizeof(int));
hipMalloc((void**)&gc,sizerowa*sizeof(int));


hipMemcpy(ga,a,sizerowa*sizeof(int),hipMemcpyHostToDevice);


	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

			do{
			numOfBlocks=current_size/blocksize;
			if( current_size%blocksize>0) numOfBlocks++;
			
			if (current_size<blocksize)
			{
				flag=1 ; 
			}	

			VectorMatrix<<<numOfBlocks,blocksize,blocksize*sizeof(int)>>>(ga,gc,current_size,gc,count_first_second);
			/*
			if (current_size%2==0)
			{
			current_size=current_size/2;
			}
			else{
			current_size=current_size/2+1;
			}
			*/
			current_size = numOfBlocks ;
			count_first_second++;	
			}while(flag!=1);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&et,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
		
		hipDeviceSynchronize();

		hipMemcpy(c,gc,1*sizeof(int),hipMemcpyDeviceToHost);
			
	printf(" parallel %f\n",et);
    

printf("%d",*(c+0));
return 0 ; 
}

